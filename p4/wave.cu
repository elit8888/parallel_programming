/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define BLOCKSIZE 256

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                     /* number of time steps */
    tpoints,                /* total points along string */
    rcode;                      /* generic return code */
float  values[MAXPOINTS];     /* values at time t */

/**********************************************************************
 *  Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0;
   tmp = tpoints - 1;
   for (j = 0; j < tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   }
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__global__
void do_math(float *values, int npoints, int niters)
{
   int myidx = blockIdx.x * BLOCKSIZE + threadIdx.x;
   if (myidx < npoints-1)
   {
       float dtime, c, dx, tau, sqtau;
       float ov, va, nv;

       dtime = 0.3;
       c = 1.0;
       dx = 1.0;
       tau = (c * dtime / dx);
       sqtau = tau * tau;
       va = values[myidx];
       ov = va;
       for (int i = 0; i < niters; i++)
       {
           nv = (2.0 * va) - ov + (sqtau * (-2.0)*va);  // Can't combine, the answer will be different.
           ov = va;
           va = nv;
       }
       values[myidx] = va;
   }
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
   float *vd;
   int tt_size = tpoints * sizeof(float);
   hipMalloc((void**)&vd, tt_size);
   hipMemcpy(vd, values, tt_size, hipMemcpyHostToDevice);

   // Determine GridSize and BlockSize
   int gridSize = (tpoints-1) / BLOCKSIZE + 1;
   //dim3 dimGrid(gridSize, 1);
   //dim3 dimBlock(BLOCKSIZE, 1);

   /* Update values for each time step */
   do_math<<<gridSize, BLOCKSIZE>>>(vd, tpoints, nsteps);
      /* Update points along line for this time step */
      /* Update old values with new values */
   hipMemcpy(values, vd, tt_size, hipMemcpyDeviceToHost);
   hipFree(vd);
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 0; i < tpoints; i++) {
      printf("%6.4f ", values[i]);
      if ((i+1)%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *  Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();
    printf("Initializing points on the line...\n");
    init_line();
    printf("Updating all points for all time steps...\n");
    update();
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");

    return 0;
}
